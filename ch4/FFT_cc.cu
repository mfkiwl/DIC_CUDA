#include "hip/hip_runtime.h"
#include"Header.h"

#include <hipfft/hipfft.h>

#include <hipfft/hipfftw.h>
#include<hip/device_functions.h>
#include <cstdlib>
#include <cstdio>
#include <cstring>

__global__ void conjugate(hipFloatComplex *comp)
{
	int tid = threadIdx.x + threadIdx.y*blockDim.x;
	int bid = blockIdx.x;
	int dim = blockDim.x*blockDim.y;
	comp[bid*dim+tid].y = -comp[bid*dim + tid].y;

}



__global__ void multiplication(hipFloatComplex *comp1, hipFloatComplex *comp2)
{
	int tid = threadIdx.x + threadIdx.y*blockDim.x;
	int bid = blockIdx.x;
	int dim = blockDim.x*blockDim.y;
	float t;
	t = comp1[bid*dim + tid].x;
	comp1[bid*dim + tid].x = comp1[bid*dim + tid].x * comp2[bid*dim + tid].x - comp1[bid*dim + tid].y * comp2[bid*dim + tid].y;
	comp1[bid*dim + tid].y = t * comp2[bid*dim + tid].y + comp1[bid*dim + tid].y * comp2[bid*dim + tid].x;
}

__global__ void latched_position(double *Mats, double *displacement_x, double *displacement_y,int subsetsize)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	int bid = blockIdx.x;
	int dim = blockDim.x*blockDim.y;
	__shared__ double max[64 * 64];
	__shared__ int a[64 * 64], b[64 * 64];
	if (x + dim*y == 0)
	{
		for (int j = 0;j < 64 * 64;j++)
		{
			max[j] = 0;
			a[j] = 0;
			b[j] = 0;
		}
	}
	max[x + blockDim.x*y] = Mats[bid*dim+x + blockDim.x*y]/subsetsize;
	int k = 64*64/ 2;
	a[x + blockDim.x*y] = x;
	b[x + blockDim.x*y] = y;
	while (k != 0)
	{
		if (max[x + blockDim.x*y] < max[x + blockDim.x*y + k])
		{
			max[x + blockDim.x*y] = max[x + blockDim.x*y + k];
			a[x + blockDim.x*y] = a[x + blockDim.x*y + k];
			b[x + blockDim.x*y] = b[x + blockDim.x*y + k];
		}
		__syncthreads();
		k = k / 2;

	}
	if (x + blockDim.x*y == 0)
	{
		if (max[0] < 0.03)
		{
			displacement_x[bid] = 0;
			displacement_y[bid] = 0;
		}
		else
		{

			displacement_x[bid] = a[0];
			displacement_y[bid] = b[0];
		}
	}
}



void CU_FFTCC(int iSubsetH, int iSubsetW, int iNumberX, int iNumberY, float *subset_aveR, float *subset_aveT, int d_iU, int d_iV)
{
	hipFloatComplex *CU_FT_R, *CU_FT_T;
	hipfftReal * CuFFT;
	int iNumbersize = iNumberX*iNumberY;
	int subsetsize = iSubsetH*iSubsetH;
	checkCudaErrors(hipMalloc((void **)&CU_FT_R, iNumbersize* iSubsetH *(iSubsetW / 2 + 1) * sizeof(hipFloatComplex)));
	checkCudaErrors(hipMalloc((void **)&CU_FT_T, iNumbersize* iSubsetH *(iSubsetW / 2 + 1) * sizeof(hipFloatComplex)));
	checkCudaErrors(hipMalloc((void **)&CuFFT, iNumbersize*iSubsetH *iSubsetW * sizeof(hipfftReal)));

	hipfftHandle plan1, plan2;
	checkCudaErrors(hipfftPlan2d(&plan1, iSubsetH, iSubsetW, HIPFFT_R2C));
	checkCudaErrors(hipfftPlan2d(&plan2, iSubsetH, iSubsetW, HIPFFT_C2R));
	dim3 block(iNumbersize);
	dim3 threadnew(iSubsetH, iSubsetW / 2 + 1);
	dim3 thread(iSubsetH, iSubsetW);
	for (int i = 0;i < iNumbersize;i++)
	{
		checkCudaErrors(hipfftExecR2C(plan1, (hipfftReal *)(subset_aveR + 1 + i * (subsetsize + 1)), (hipFloatComplex *)(CU_FT_R + i*subsetsize)));
		checkCudaErrors(hipfftExecR2C(plan1, (hipfftReal *)(subset_aveT + 1 + i * (subsetsize + 1)), (hipFloatComplex *)(CU_FT_T + i*subsetsize)));
	}
	conjugate << <block, threadnew >> > (CU_FT_R);
	multiplication << <block, threadnew >> > (CU_FT_R, CU_FT_T);

	for (int j = 0;j < iNumbersize;j++)
	{
		checkCudaErrors(hipfftExecC2R(plan2, (hipFloatComplex *)CU_FT_R, (hipfftReal *)(CuFFT + subsetsize)));
	}


	latched_position << <block, thread2 >> > (CuFFT, d_iU, d_iV, subsetsize);
}
